
#include <hip/hip_runtime.h>
#ifdef OCTOTIGER_HAVE_CUDA
#include "octotiger/common_kernel/interaction_constants.hpp"
#include "octotiger/monopole_interactions/monopole_kernel_templates.hpp"
#include "octotiger/monopole_interactions/p2p_cuda_kernel.hpp"

namespace octotiger {
namespace fmm {
    namespace monopole_interactions {
        // __constant__ octotiger::fmm::multiindex<> device_stencil_const[P2P_PADDED_STENCIL_SIZE];
        __device__ __constant__ float device_stencil_masks[FULL_STENCIL_SIZE];
        __device__ __constant__ double device_four_constants[FULL_STENCIL_SIZE * 4];

        //__device__ const size_t component_length = ENTRIES + SOA_PADDING;
        __device__ const size_t component_length_unpadded = INNER_CELLS + SOA_PADDING;
        __device__ const size_t cache_line_length = INX + 10;
        __device__ const size_t cache_offset = INX + STENCIL_MIN;

        __global__ void
        __launch_bounds__(INX * INX, 4)
        cuda_p2p_interactions_kernel(
            const double (&local_monopoles)[NUMBER_LOCAL_MONOPOLE_VALUES],
            double (&potential_expansions)[NUMBER_POT_EXPANSIONS_SMALL],
            const double theta, const double dx) {
            __shared__ double monopole_cache[cache_line_length * cache_line_length];
            __shared__ multiindex<> coarse_index_cache[cache_line_length * cache_line_length];
            // get local id
            int local_id = threadIdx.y * INX + threadIdx.z;

            // use in case of debug prints
            //bool first_thread = (blockIdx.x == 0) && (threadIdx.y == 0) && (threadIdx.z == 0);
            // Set cell indices
            const octotiger::fmm::multiindex<> cell_index((threadIdx.x + blockIdx.x * 1) + INNER_CELLS_PADDING_DEPTH,
                threadIdx.y + INNER_CELLS_PADDING_DEPTH, threadIdx.z + INNER_CELLS_PADDING_DEPTH);
            octotiger::fmm::multiindex<> cell_index_coarse(cell_index);
            cell_index_coarse.transform_coarse();
            const size_t cell_flat_index = octotiger::fmm::to_flat_index_padded(cell_index);
            octotiger::fmm::multiindex<> cell_index_unpadded((threadIdx.x + blockIdx.x * 1), threadIdx.y, threadIdx.z);
            const size_t cell_flat_index_unpadded =
                octotiger::fmm::to_inner_flat_index_not_padded(cell_index_unpadded);

            // Required for mask
            const double theta_rec_squared = sqr(1.0 / theta);
            const double d_components[2] = {1.0 / dx, -1.0 / dx};
            double tmpstore[4] = {0.0, 0.0, 0.0, 0.0};

            const size_t index_base = (threadIdx.y + 5) * (INX + 10) + threadIdx.z + 5;
            int load_offset = 0;
            int load_id = local_id;
            if (local_id >= cache_line_length) {
              load_offset = 1;
              load_id = load_id - cache_line_length;
            }

            for (int stencil_x = STENCIL_MIN; stencil_x <= STENCIL_MAX; stencil_x++) {
                int x = stencil_x - STENCIL_MIN;
                __syncthreads();
                if (local_id < cache_line_length * 2) {
                    for (int i = 0; i < cache_line_length / 2; i++) {
                        const multiindex<> partner_index(INNER_CELLS_PADDING_DEPTH + blockIdx.x + stencil_x,
                                                            2*i + load_offset + cache_offset,
                                                            cache_offset + load_id);
                        const size_t partner_flat_index = to_flat_index_padded(partner_index);
                        multiindex<> partner_index_coarse(partner_index);
                        partner_index_coarse.transform_coarse();
                        coarse_index_cache[cache_line_length*(2*i + load_offset) + load_id] = partner_index_coarse;
                        monopole_cache[cache_line_length*(2*i + load_offset) + load_id] = local_monopoles[partner_flat_index];

                    }
                }
                __syncthreads();
                for (int stencil_y = STENCIL_MIN; stencil_y <= STENCIL_MAX; stencil_y++) {
                    int y = stencil_y - STENCIL_MIN;
                    for (int stencil_z = STENCIL_MIN; stencil_z <= STENCIL_MAX; stencil_z++) {
                        const size_t index = x * STENCIL_INX * STENCIL_INX + y * STENCIL_INX + (stencil_z - STENCIL_MIN);
                        if (!device_stencil_masks[index]) {
                            continue;
                        }
                        const size_t cache_index = index_base + stencil_y * cache_line_length +
                stencil_z;

                        // Create mask
                        const double theta_c_rec_squared = static_cast<double>(
                            distance_squared_reciprocal(cell_index_coarse, coarse_index_cache[cache_index]));
                        const bool mask_b = theta_rec_squared > theta_c_rec_squared;
                        double mask = mask_b ? 1.0 : 0.0;

                        const double monopole = monopole_cache[cache_index] * mask * d_components[0];
                        const double four[4] = {device_four_constants[index * 4 + 0],
                                                device_four_constants[index * 4 + 1],
                                                device_four_constants[index * 4 + 2],
                                                device_four_constants[index * 4 + 3]};
                        tmpstore[0] = tmpstore[0] + four[0] * monopole;
                        tmpstore[1] = tmpstore[1] + four[1] * monopole * d_components[1];
                        tmpstore[2] = tmpstore[2] + four[2] * monopole * d_components[1];
                        tmpstore[3] = tmpstore[3] + four[3] * monopole * d_components[1];
                        // compute_monopole_interaction<double>(monopole, tmpstore, four, d_components);
                    }
                }
            }

            // Store results in output arrays
            potential_expansions[cell_flat_index_unpadded] = tmpstore[0];
            potential_expansions[1 * component_length_unpadded +
                cell_flat_index_unpadded] = tmpstore[1];
            potential_expansions[2 * component_length_unpadded +
                cell_flat_index_unpadded] = tmpstore[2];
            potential_expansions[3 * component_length_unpadded +
                cell_flat_index_unpadded] = tmpstore[3];
        }
    }    // namespace monopole_interactions
}    // namespace fmm
}    // namespace octotiger
#endif
